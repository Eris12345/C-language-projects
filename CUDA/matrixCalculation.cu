#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
using namespace std;
/* initializes the given matrix as a NULL (ZERO) matrix
* (In the NULL (ZERO) matrix all elements are 0) */
__global__ void initNullGPU(
	int *devMatrix, int numR, int numC) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = x * gridDim.y + y;
	if (x < numR && y < numC)
		devMatrix[offset] = 0;
}
/* initializes all the elements of the given matrix with the given value */
__global__ void initValueGPU(
	int *devMatrix, int numR, int numC, int value) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = x * gridDim.y + y;
	if (x < numR && y < numC)
		devMatrix[offset] = value;
}
/* initializes the given square matrix as an IDENTITY matrix
* (In the IDENTITY matrix all diagonal elements are 1) */
__global__ void initIdentityGPU(
	int *devMatrix, int numR, int numC) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = x * gridDim.y + y;
	if (x < numR && y < numC)
		if (x == y)
			devMatrix[offset] = 1;
		else
			devMatrix[offset] = 0;
}
/* initializes the upper triangular matrix with the given value
* (In the UPPER TRIANGULAR matrix all the elements in the upper
* triangular segment have the same value) */
__global__ void initUpperTriangularValueGPU(
	int *devMatrix, int numR, int numC, int value) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = x * gridDim.y + y;
	if (x < numR && y < numC)
		if (x > y)
			devMatrix[offset] = 0;
		else
			devMatrix[offset] = value;
}
/* initializes the given scalar matrix with the given value
* (In the SCALAR matrix all diagonal elements have the same value) */
__global__ void initScalarValueGPU(
	int *devMatrix, int numR, int numC, int value) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = x * gridDim.y + y;

	if (x < numR && y < numC)
		if (x == y)
			devMatrix[offset] = value;
		else
			devMatrix[offset] = 0;
}
/* adds the given left and right matrices onto the result matrix. */
__global__ void addGPU(
	int *devMatrixLeft, int *devMatrixRight, int *devMatrixResult,
	int numR, int numC) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = x * gridDim.y + y;
	if (x < numR && y < numC)
		devMatrixResult[offset] = devMatrixLeft[offset] + devMatrixRight[offset];
}
/* calculates and returns the transpose of the given matrix. */
__global__ void transposeGPU(
	int *devMatrix, int *devTranspose,
	int numR, int numC) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offsetMatrix = x * gridDim.y + y;
	int offsetTranspose = y * gridDim.x + x;
	if (x < numR && y < numC)
		devTranspose[offsetTranspose] = devMatrix[offsetMatrix];
}
class Matrix {
	int *hostMatrix; //matrix as a one-dimensional array
	int numRows,
		numCols;
public:
	Matrix(int numR, int numC);
	Matrix(const Matrix& m);
		Matrix& operator=(const Matrix& right);
	~Matrix();
	friend ostream& operator<<(ostream& out, const Matrix& right);
	//member functions to initialize the matrix:
	void initNull();
	void initValue(int value);
	void initIdentity();
	void initUpperTriangularValue(int value);
	void initScalarValue(int value);
	Matrix operator+(const Matrix& right);
	friend Matrix transpose(const Matrix& matrix);
};

Matrix::Matrix(int numR, int numC) {
	numRows = numR;
	numCols = numC;
	hostMatrix = new int[numRows * numCols];
}

	Matrix::Matrix(const Matrix& m) {
	numRows = m.numRows;
	numCols = m.numCols;
	hostMatrix = new int[numRows * numCols];
	memcpy(hostMatrix, m.hostMatrix, sizeof(int) * numRows * numCols);
}
  Matrix& Matrix :: operator=(const Matrix& right) {
	  if (this != &right) {
		  delete[] hostMatrix;
		  numRows = right.numRows;
		  numCols = right.numCols;
		  hostMatrix = new int[numRows * numCols];
		  memcpy(hostMatrix, right.hostMatrix, sizeof(int) * numRows * numCols);
	  }
	  return (*this);
  }
  Matrix :: ~Matrix() {
	  delete[] hostMatrix;
  }
  ostream& operator<<(ostream& out, const Matrix& right) {
	  int r, c, offset = 0;
	  out << "A " << right.numRows << " by " << right.numCols << " matrix:\n";
	  for (r = 0; r < right.numRows; r++) {
		  out << "| ";
		  for (c = 0; c < right.numCols - 1; c++)
			  out << right.hostMatrix[offset++] << " ";
		  out << right.hostMatrix[offset++] << " |\n";
	  }
	  return out;
  }
  /* initializes the given matrix as a NULL (ZERO) matrix
  * (In the NULL (ZERO) matrix all elements are 0) */
  void Matrix::initNull() {
	  int *devMatrix;
	  hipMalloc((void**)&devMatrix, sizeof(int) * numRows * numCols);
	  dim3 grid(numRows, numCols);
	  initNullGPU << <grid, 1 >> > (devMatrix, numRows, numCols);
	  hipMemcpy(hostMatrix, devMatrix,
		  sizeof(int) * numRows * numCols, hipMemcpyDeviceToHost);
	  hipFree(devMatrix);
  }
  /* initializes all the elements of the given matrix with the given value */
  void Matrix::initValue(int value) {
	  int *devMatrix;
	  hipMalloc((void**)&devMatrix, sizeof(int) * numRows * numCols);
	  dim3 grid(numRows, numCols);
	  initValueGPU << <grid, 1 >> > (devMatrix, numRows, numCols, value);
	  hipMemcpy(hostMatrix, devMatrix,
		  sizeof(int) * numRows * numCols, hipMemcpyDeviceToHost);
	  hipFree(devMatrix);
  }

	/* initializes the given square matrix as an IDENTITY matrix
	* (In the IDENTITY matrix all diagonal elements are 1) */
	void Matrix::initIdentity() {
	int *devMatrix;
	hipMalloc((void**)&devMatrix, sizeof(int) * numRows * numCols);
	dim3 grid(numRows, numCols);
	initIdentityGPU << <grid, 1 >> > (devMatrix, numRows, numCols);
	hipMemcpy(hostMatrix, devMatrix,
		sizeof(int) * numRows * numCols, hipMemcpyDeviceToHost);
	hipFree(devMatrix);
}
  /* initializes the upper triangular matrix with the given value
  * (In the UPPER TRIANGULAR matrix all the elements in the upper
  * triangular segment have the same value) */
  void Matrix::initUpperTriangularValue(int value) {
	  int *devMatrix;
	  hipMalloc((void**)&devMatrix, sizeof(int) * numRows * numCols);
	  dim3 grid(numRows, numCols);
	  initUpperTriangularValueGPU << <grid, 1 >> > (devMatrix, numRows, numCols, value);
	  hipMemcpy(hostMatrix, devMatrix,
		  sizeof(int) * numRows * numCols, hipMemcpyDeviceToHost);
	  hipFree(devMatrix);
  }
  /* initializes the given scalar matrix with the given value
  * (In the SCALAR matrix all diagonal elements have the same value) */
  void Matrix::initScalarValue(int value) {
	  int *devMatrix;
	  hipMalloc((void**)&devMatrix, sizeof(int) * numRows * numCols);
	  dim3 grid(numRows, numCols);
	  initScalarValueGPU << <grid, 1 >> > (devMatrix, numRows, numCols, value);
	  hipMemcpy(hostMatrix, devMatrix,
		  sizeof(int) * numRows * numCols, hipMemcpyDeviceToHost);
	  hipFree(devMatrix);
  }
  Matrix Matrix :: operator+(const Matrix& right) {
	  if (numRows != right.numRows ||
		  numCols != right.numCols) {
		  cerr << "Error: invalid sizes in matrix addition.\n";
		  cerr << "Left matrix object is returned.\n";
		  return (*this);
	  }
	  Matrix result(numRows, numCols);
	  int *devMatrixLeft,
		  *devMatrixRight,
		  *devMatrixResult;
	  hipMalloc((void**)&devMatrixLeft, sizeof(int) * numRows * numCols);
	  hipMalloc((void**)&devMatrixRight, sizeof(int) * numRows * numCols);
	  hipMalloc((void**)&devMatrixResult, sizeof(int) * numRows * numCols);
	  hipMemcpy(devMatrixLeft, hostMatrix,
		  sizeof(int) * numRows * numCols, hipMemcpyHostToDevice);
	  hipMemcpy(devMatrixRight, right.hostMatrix,
		  sizeof(int) * numRows * numCols, hipMemcpyHostToDevice);
 
	  dim3 grid(numRows, numCols);
	addGPU << <grid, 1 >> > (devMatrixLeft, devMatrixRight, devMatrixResult,
		numRows, numCols);
	hipMemcpy(result.hostMatrix, devMatrixResult,
		sizeof(int) * numRows * numCols, hipMemcpyDeviceToHost);
	hipFree(devMatrixLeft);
	hipFree(devMatrixRight);
	hipFree(devMatrixResult);
	return result;
  }
  /* calculates and returns the transpose of the given matrix. */
  Matrix transpose(const Matrix& matrix) {
	  int numRows = matrix.numRows;
	  int numCols = matrix.numCols;
	  Matrix result(numCols, numRows);
	  int *devMatrix, *devTranspose;
	  hipMalloc((void**)&devMatrix, sizeof(int) * numRows * numCols);
	  hipMalloc((void**)&devTranspose, sizeof(int) * numRows * numCols);
	  hipMemcpy(devMatrix, matrix.hostMatrix,
		  sizeof(int) * numRows * numCols, hipMemcpyHostToDevice);
	  dim3 grid(numRows, numCols);
	  transposeGPU << <grid, 1 >> > (devMatrix, devTranspose, numRows, numCols);
	  hipMemcpy(result.hostMatrix, devTranspose,
		  sizeof(int) * numRows * numCols, hipMemcpyDeviceToHost);
	  hipFree(devMatrix);
	  hipFree(devTranspose);
	  return result;
  }
  int main(void) {
	  Matrix a(3, 5), b(3, 5), c(3, 5), d(5, 3), e(4, 4);
	  a.initValue(2);
	  b.initValue(3);
	  cout << "a: \n" << a << endl;
	  cout << "b: \n" << b << endl;
	  c = a + b;
	  cout << "a + b = c: \n" << c << endl;
	  d.initNull();
	  cout << "null matrix d: \n" << d << endl;
	  e.initIdentity();
	  cout << "identity matrix e: \n" << e << endl;
	  e.initUpperTriangularValue(7);
	  cout << "upper triangular matrix e: \n" << e << endl;
	  e = transpose(e);
	  cout << "transpose of e: \n" << e << endl;
	  e.initScalarValue(9);
	  cout << "scalar matrix e: \n" << e << endl;
	  return 0;
  }
